#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");

  // check batch has finished copying to the device
  CUDA_CHECK(hipStreamWaitEvent(hipStreamDefault, batch->copied_, 0));

  // Reshape to loaded data.
  if (this->transform_param_.use_gpu_transform()) {
    // instead of copy, perform out-of-place transform(!)
    this->data_transformer_->TransformGPU(top[0]->num(),
                                       top[0]->channels(),
                                       batch->data_.height(),
                                       batch->data_.width(),
                                       batch->data_.gpu_data(),
                                       top[0]->mutable_gpu_data(),
                                       batch->random_vec_.mutable_gpu_data());
  }  else {
    // Copy the data
    // Reshape to loaded data.
    top[0]->ReshapeLike(batch->data_);
    caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
               top[0]->mutable_gpu_data());
  }

  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

}  // namespace caffe
